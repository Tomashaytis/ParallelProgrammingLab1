﻿#include <iostream>
#include <cstdlib>
#include <chrono>
#include <fstream>
#include <string>
// �� ����, ����� �� ���
#include "hip/hip_runtime.h"

//

#define BLOCK_SIZE 20

__global__ void matrix_dot(int* A, int* B, int* C, int size)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = blockIdx.x;
    int ty = blockIdx.y;
    int sum = 0;
    int ia = size * BLOCK_SIZE * by + size * ty;
    int ib = BLOCK_SIZE * bx + tx;
    for (int k = 0; k < size; k++)
        sum += A[ia + k] * B[ib + k * size];
    int ic = size * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[ic + size * ty + tx] = sum;
}

class SquareMatrix
{
private:
    int* _data;
    size_t _size;

public:
    SquareMatrix(size_t initial_size = 10)
    {
        _size = initial_size;
        _data = new int[_size * _size];
    }
    ~SquareMatrix()
    {
        clear();
    }
    SquareMatrix(const SquareMatrix& src)
    {
        *this = src;
    }
    SquareMatrix& operator=(const SquareMatrix& src)
    {
        clear();
        _size = src._size;
        _data = new int[_size * _size];
        for (size_t i = 0; i < _size * _size; i++)
        {
            _data[i] = src._data[i];
        }
        return *this;
    }
    bool operator==(const SquareMatrix& rhs) const
    {
        if (_size != rhs._size)
            return false;
        for (size_t i = 0; i < _size; i++)
        {
            if (_data[i] != rhs._data[i])
                return false;
        }
        return true;
    }
    int& operator()(size_t i, size_t j)
    {
        return _data[i * _size + j];
    }
    int operator()(size_t i, size_t j) const
    {
        return _data[i * _size + j];
    }
    void clear()
    {
        delete[] _data;
        _data = NULL;
        _size = 0;
    }
    size_t size() const
    {
        return _size;
    }
    void random_fill()
    {
        for (size_t i = 0; i < _size * _size; i++)
        {
            int k = rand() % 2;
            (k == 0) ? (k = -1) : (k = 1);
            _data[i] = rand() % 100 * k;
        }
    }
    void fill(int value)
    {
        for (size_t i = 0; i < _size * _size; i++)
        {
            _data[i] = value;
        }
    }
    double dot(const SquareMatrix& rhs)
    {
        std::chrono::steady_clock::time_point start, end;
        start = std::chrono::steady_clock::now();
        SquareMatrix result = SquareMatrix(_size);
        for (size_t i = 0; i < _size; i++)
        {
            for (size_t j = 0; j < _size; j++)
            {
                for (size_t k = 0; k < _size; k++)
                {
                    result(i, j) += (*this)(i, k) * rhs(k, j);
                }
            }
        }
        end = std::chrono::steady_clock::now();
        return std::chrono::duration <double>(end - start).count();
    }
    double cuda_dot(SquareMatrix& rhs)
    {
        SquareMatrix result = SquareMatrix(_size);
        int* A = _data;
        int* B = rhs._data;
        int* C = result._data;

        int* Adev = NULL;
        int* Bdev = NULL;
        int* Cdev = NULL;

        hipMalloc((void**)&Adev, _size * _size * sizeof(int));
        hipMalloc((void**)&Bdev, _size * _size * sizeof(int));
        hipMalloc((void**)&Cdev, _size * _size * sizeof(int));

        dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
        dim3 blocks(_size / threads.x, _size / threads.y);

        hipEvent_t begin, end;
        float time = 0;
        hipEventCreate(&begin);
        hipEventCreate(&end);

        hipEventRecord(begin, 0);
        hipMemcpy(A, Adev, _size * _size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(B, Bdev, _size * _size * sizeof(int), hipMemcpyHostToDevice);
        matrix_dot <<<blocks, threads>>> (Adev, Bdev, Cdev, _size);
        hipMemcpy(C, Cdev, _size * _size * sizeof(int), hipMemcpyDeviceToHost);
        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(&time, begin, end);

        hipEventDestroy(begin);
        hipEventDestroy(end);
        hipFree(Adev);
        hipFree(Bdev);
        hipFree(Cdev);
        return time;
    }
    friend std::ostream& operator<<(std::ostream& os, const SquareMatrix& data)
    {
        for (size_t i = 0; i < data.size(); i++)
        {
            for (size_t j = 0; j < data.size(); j++)
            {
                os << data(i, j) << '\t';
            }
            os << '\n';
        }
        return os;
    }
};

int main(int argc, char** argv)
{
    srand(time(0));
    int size = 1000;
    int block_sizes[] = {4, 8, 12, 16, 32, 48, 96 };
    std::string filename;
    for (int i = 0; i < 7; ++i) {
        filename = std::to_string(block_sizes[i]);
        filename += ".txt";
        std::ofstream file(filename);
#ifdef BLOCK_SIZE
#undef BLOCK_SIZE
#define BLOCK_SIZE block_sizes[i]
#endif
        for (int j = 1; j < 11; ++j) {
            SquareMatrix matrix1(j * 96), matrix2(j * 96), matrix3(j * 96);
            for (int k = 0; k < 10; ++k) {
                matrix1.random_fill();
                matrix2.random_fill();
                matrix3 = matrix1;
                file << matrix1.cuda_dot(matrix2) << " ";
                matrix3.dot(matrix2);
                if (!(matrix1 == matrix3)) {
                    std::cout << "Matrices are not equal" << std::endl;
                    return 1;
                }
            }
        }
        file.close();
    }
    return 0;
}